#include "hip/hip_runtime.h"
#include <cstdio>
#include "tree_cuda.cuh"

__device__ 
void cudaSearch(DeviceNode *node, Side side, Side maximizer, int depth) {
    if (depth == 0) {
       node->setAlpha(node->getScore());
       node->setBeta(node->getScore());
       return;
    }
    DeviceBoard *board = node->getBoard();
    Side oppositeSide = side == BLACK ? WHITE : BLACK;

    for (int i = 0; i < BOARD_SIZE; i++) {
        for (int j = 0; j < BOARD_SIZE; j++) {
            Move *move = new Move(i, j);
            if (board->checkMove(move, oppositeSide)) {
                char *black;
                char *taken;

                black = (char *) malloc(BOARD_SIZE * BOARD_SIZE * sizeof(char));
                taken = (char *) malloc(BOARD_SIZE * BOARD_SIZE * sizeof(char));

                for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; i++) {
                    black[i] = board->black[i];
                    taken[i] = board->taken[i];
                }
                DeviceBoard *newBoard = new DeviceBoard(black, taken);
                newBoard->doMove(move, oppositeSide);
                DeviceNode *child = new DeviceNode(move, oppositeSide, maximizer, newBoard);

                // pass alpha and beta values down
                child->setAlpha(node->getAlpha());
                child->setBeta(node->getBeta());

                // search child
                cudaSearch(child, oppositeSide, maximizer, depth - 1);

                if (side == maximizer) {
                    node->setBeta(min(node->getBeta(), child->getAlpha()));
                } else {
                    node->setAlpha(max(node->getAlpha(), child->getBeta()));
                }

                delete child;

                if (node->getAlpha() >= node->getBeta()) {
                    return;
                }
            }
        }
    }

    // thrust::device_vector<Move> moves = board->getMoves(oppositeSide);
    // for (int i = 0; i < moves.size(); i++) {
    //     // create the next child
    //     Move *move = new Move(moves[i].getX(), moves[i].getY());
    //     DeviceBoard *newBoard = board->copy();
    //     newBoard->doMove(move, oppositeSide);
    //     DeviceNode *child = new DeviceNode(move, oppositeSide, maximizer, newBoard);

    //     // pass alpha and beta values down
    //     child->setAlpha(node->getAlpha());
    //     child->setBeta(node->getBeta());

    //     // search child
    //     cudaSearch(child, oppositeSide, maximizer, depth - 1);

    //     if (side == maximizer) {
    //         node->setBeta(min(node->getBeta(), child->getAlpha()));
    //     } else {
    //         node->setAlpha(max(node->getAlpha(), child->getBeta()));
    //     }

    //     delete child;

    //     if (node->getAlpha() >= node->getBeta()) {
    //         return;
    //     }
    // }
}

__global__
void cudaTreeKernel(Move *moves, char *black, char *taken, int *values, Side side, 
    Side maximizer, int alpha, int beta, int depth) {
    // only one thread does high-level tasks
    if (threadIdx.x == 0) {
        // make one new node per block
        Move *move = new Move(moves[blockIdx.x].getX(), moves[blockIdx.x].getY());

        DeviceBoard *newBoard = new DeviceBoard(black, taken);
        newBoard->doMove(move, side);
        DeviceNode *node = new DeviceNode(move, side, maximizer, newBoard);

        // pass down alpha and beta
        node->setAlpha(alpha);
        node->setBeta(beta);

        cudaSearch(node, side, maximizer, depth);

        // update the values we care about - if the parent node is a maximizing node, 
        // it cares about the child alpha values
        if (side == maximizer) {
            values[blockIdx.x] = node->getBeta();
        } else {
            values[blockIdx.x] = node->getAlpha();
        }
        printf("values: %d index: %d\n", values[blockIdx.x], blockIdx.x);
    }

}

void cudaCallTreeKernel(Move *moves, char *black, char *taken, int *values, Side side, 
    Side maximizer, int alpha, int beta, int numMoves, int depth) {

    printf("hello...\n");

    cudaTreeKernel<<<numMoves, 64>>>(moves, black, taken, values, side, 
       maximizer, alpha, beta, depth);
}
